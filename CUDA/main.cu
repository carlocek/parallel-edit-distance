#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <string>
#include <cstring>
#include <cmath>
#include <algorithm>
#include <chrono>

#define MIN(x,y) ((x) < (y) ? (x) : (y))
#define MAX(x,y) ((x) < (y) ? (y) : (x))

using namespace std;

char* generateRandString(char* str, int size)
{
	const int ch_MAX = 4;
//  char alphabet[ch_MAX] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g',
//                          'h', 'i', 'j', 'k', 'l', 'm', 'n',
//                          'o', 'p', 'q', 'r', 's', 't', 'u',
//                          'v', 'w', 'x', 'y', 'z' };
	char alphabet[ch_MAX] = {'a', 'b', 'c', 'd'};
    for(int i = 0; i < size; i++)
        str[i] = alphabet[rand() % ch_MAX];
    str[size] = '\0';
    return str;
}

__global__ void editDistKernel(char* devA, char* devB, int lenA, int lenB, unsigned int* d_prevprevDiagPtr, unsigned int* d_prevDiagPtr, unsigned int* d_currDiagPtr, int d)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int bd = blockDim.x;
	int j = lenA+d-tid;

	if(d < 1)
	{
		if(tid == 0)
			d_currDiagPtr[0] = MIN(lenA+d, lenB+1);
		else if(tid == lenA+d)
			d_currDiagPtr[lenA+d] = MIN(lenA+d, lenB+1);
	}
	if(tid < MIN(lenA+d, lenB+1) && tid > MAX(0, d-1))
	{
		if(devA[j-1] != devB[tid-1])
			d_currDiagPtr[tid] = 1 + MIN(d_prevDiagPtr[tid], MIN(d_prevDiagPtr[tid-1], d_prevprevDiagPtr[tid-1]));
		else
			d_currDiagPtr[tid] = d_prevprevDiagPtr[tid-1];
	}
}

int main()
{
	for(int it = 0; it < 20; it++)
	{
	chrono::high_resolution_clock::time_point t1, t2;
	chrono::duration<double> time_span;

	int lenA = 10000;
	int lenB = 10000;
	char* A = new char[lenA+1];
	char* B = new char[lenB+1];
//	char A[lenA] = "sundayy";
//	char B[lenB] = "sfnlayz";
	char* devA;
	char* devB;
	hipMalloc((void**)&devA, (lenA+1)*sizeof(char));
	hipMalloc((void**)&devB, (lenB+1)*sizeof(char));
	generateRandString(A, lenA);
	generateRandString(B, lenB);
	hipMemcpy((void*)devA, (void*)A, (lenA+1)*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy((void*)devB, (void*)B, (lenB+1)*sizeof(char), hipMemcpyHostToDevice);

	t1 = chrono::high_resolution_clock::now();

	unsigned int* d_currDiag;
	unsigned int* d_prevDiag;
	unsigned int* d_prevprevDiag;

	unsigned int* d_currDiagPtr;
	unsigned int* d_prevDiagPtr;
	unsigned int* d_prevprevDiagPtr;

	unsigned int* currDiag = new unsigned int[lenA+1];
	unsigned int* prevDiag = new unsigned int[lenA+1];
	unsigned int* prevprevDiag = new unsigned int[lenA+1];
	hipMalloc((void**)&d_currDiag, (lenA+1)*sizeof(unsigned int));
	hipMalloc((void**)&d_prevDiag, (lenA+1)*sizeof(unsigned int));
	hipMalloc((void**)&d_prevprevDiag, (lenA+1)*sizeof(unsigned int));

	prevprevDiag[0] = 0;
	prevDiag[0] = 1;
	prevDiag[1] = 1;
	hipMemcpy((void*)d_prevprevDiag, (void*)prevprevDiag, (lenA+1)*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy((void*)d_prevDiag, (void*)prevDiag, (lenA+1)*sizeof(unsigned int), hipMemcpyHostToDevice);

	d_prevprevDiagPtr = d_prevprevDiag;
	d_prevDiagPtr = d_prevDiag;
	d_currDiagPtr = d_currDiag;

//	int bDim = 128;
//	int gDim = ceil((float)(lenA+1)/bDim);
	int dmin = 2-lenA;
	int dmax = lenB+1;
	int ed;
	for(int d = dmin; d < dmax; d++)
	{
		int imax = min(lenA+d+1, lenB+1);
		int bDim = min(imax, 512);
		int gDim = ceil((float)(imax)/bDim);
		editDistKernel<<<gDim, bDim>>>(devA, devB, lenA, lenB, d_prevprevDiagPtr, d_prevDiagPtr, d_currDiagPtr, d);

		unsigned int* tmp = d_prevprevDiagPtr;
		d_prevprevDiagPtr = d_prevDiagPtr;
		d_prevDiagPtr = d_currDiagPtr;
		d_currDiagPtr = tmp;
	}
	hipDeviceSynchronize();
	hipMemcpy((void*)&ed, (void*)&d_prevDiag[lenA], 1*sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	t2 = chrono::high_resolution_clock::now();
	hipFree(devA);
	hipFree(devB);
	hipFree(d_prevprevDiagPtr);
	hipFree(d_prevDiagPtr);
	hipFree(d_currDiagPtr);
	time_span = chrono::duration_cast<chrono::duration<double>>(t2 - t1);
	printf("\nelapsed time: %f\n", time_span.count());
	printf("edit distance: %d\n", ed);
	ed = 0;
	}
	return 0;
}
