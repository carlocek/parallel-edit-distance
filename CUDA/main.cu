#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <cstring>
#include <cmath>
#include <algorithm>
#include <chrono>

#define MIN(x,y) ((x) < (y) ? (x) : (y))

using namespace std;

char* generateRandString(char* str, int size)
{
	const int ch_MAX = 4;
//  char alphabet[ch_MAX] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g',
//                          'h', 'i', 'j', 'k', 'l', 'm', 'n',
//                          'o', 'p', 'q', 'r', 's', 't', 'u',
//                          'v', 'w', 'x', 'y', 'z' };
	char alphabet[ch_MAX] = {'a', 'b', 'c', 'd'};
    for(int i = 0; i < size; i++)
        str[i] = alphabet[rand() % ch_MAX];
    str[size] = '\0';
    return str;
}

__global__ void editDistKernel(char* devA, char* devB, int lenA, int lenB, unsigned int* d_prevprevDiagPtr, unsigned int* d_prevDiagPtr, unsigned int* d_currDiagPtr, int d)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int bd = blockDim.x;
	int j = lenA+d-tid;
	__shared__ extern unsigned int ds_currDiag[];

	if(d < 1)
	{
		if(tid == 0)
			ds_currDiag[0] = min(lenA+d, lenB+1);
		else if(tid == lenA+d)
			ds_currDiag[(lenA+d) % bd] = min(lenA+d, lenB+1);
	}
	if(tid < min(lenA+d, lenB+1) && tid > max(0, d-1))
	{
		if(devA[j-1] != devB[tid-1])
			ds_currDiag[tid % bd] = 1 + MIN(d_prevDiagPtr[tid], MIN(d_prevDiagPtr[tid-1], d_prevprevDiagPtr[tid-1]));
		else
			ds_currDiag[tid % bd] = d_prevprevDiagPtr[tid-1];
	}
//	__threadfence();
	__syncthreads();
	if(tid < lenA+1)
	{
		d_currDiagPtr[tid] = ds_currDiag[tid % bd];
	}

//	__threadfence();
//	if(tid % blockDim.x == 0)
//	{
//		unsigned int* tmp = d_prevprevDiagPtr;
//		d_prevprevDiagPtr = d_prevDiagPtr;
//		d_prevDiagPtr = d_currDiagPtr;
//		d_currDiagPtr = tmp;
//	}
//	__threadfence();
//	if(tid == 0)
//	{
//		for(int i = 0; i < lenA+1; i++)
//			printf("%d, ", d_currDiagPtr[i]);
//		printf("\n");
//	}
}

int main()
{
	for(int it = 0; it < 20; it++)
	{
	chrono::high_resolution_clock::time_point t1, t2;
	chrono::duration<double> time_span;

	int lenA = 10000;
	int lenB = 10000;
	char* A = new char[lenA+1];
	char* B = new char[lenB+1];
//	char A[lenA] = "sundayy";
//	char B[lenB] = "sfnlayz";
	char* devA;
	char* devB;
	hipMalloc((void**)&devA, (lenA+1)*sizeof(char));
	hipMalloc((void**)&devB, (lenB+1)*sizeof(char));
	generateRandString(A, lenA);
	generateRandString(B, lenB);
	hipMemcpy((void*)devA, (void*)A, (lenA+1)*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy((void*)devB, (void*)B, (lenB+1)*sizeof(char), hipMemcpyHostToDevice);

	for(int i = 0; i < lenA; i++)
		printf("%c", A[i]);
	printf("\n");
	for(int i = 0; i < lenB; i++)
		printf("%c", B[i]);
	printf("\n");

	t1 = chrono::high_resolution_clock::now();

	unsigned int* d_currDiag;
	unsigned int* d_prevDiag;
	unsigned int* d_prevprevDiag;

	unsigned int* d_currDiagPtr;
	unsigned int* d_prevDiagPtr;
	unsigned int* d_prevprevDiagPtr;

	unsigned int* currDiag = new unsigned int[lenA+1];
	unsigned int* prevDiag = new unsigned int[lenA+1];
	unsigned int* prevprevDiag = new unsigned int[lenA+1];
	hipMalloc((void**)&d_currDiag, (lenA+1)*sizeof(unsigned int));
	hipMalloc((void**)&d_prevDiag, (lenA+1)*sizeof(unsigned int));
	hipMalloc((void**)&d_prevprevDiag, (lenA+1)*sizeof(unsigned int));

	prevprevDiag[0] = 0;
	prevDiag[0] = 1;
	prevDiag[1] = 1;
	hipMemcpy((void*)d_prevprevDiag, (void*)prevprevDiag, (lenA+1)*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy((void*)d_prevDiag, (void*)prevDiag, (lenA+1)*sizeof(unsigned int), hipMemcpyHostToDevice);
//	cudaMemcpy((void*)d_currDiag, (void*)currDiag, (lenA+1)*sizeof(unsigned int), cudaMemcpyHostToDevice);

	d_prevprevDiagPtr = d_prevprevDiag;
	d_prevDiagPtr = d_prevDiag;
	d_currDiagPtr = d_currDiag;

	int bDim = 256;
	int gDim = ceil((float)(lenA+1)/bDim);
	int dmin = 2-lenA;
	int dmax = lenB+1;
	int ed;
	for(int d = dmin; d < dmax; d++)
	{
//		int imax = min(lenA+d+1, lenB+1);
//		int gDim = ceil((float)(imax)/bDim);
		editDistKernel<<<gDim, bDim, (bDim)*sizeof(unsigned int)>>>(devA, devB, lenA, lenB, d_prevprevDiagPtr, d_prevDiagPtr, d_currDiagPtr, d);
		hipDeviceSynchronize();

//		cudaMemcpy((void*)currDiag, (void*)d_currDiagPtr, (lenA+1)*sizeof(unsigned int), cudaMemcpyDeviceToHost);
//		for(int i = 0; i < lenA+1; i++)
//			printf("%d, ", currDiag[i]);
//		printf("\n");
//		printf("end of iteration\n");
		if(d == dmax-1)
		{
			hipMemcpy((void*)&ed, (void*)&d_currDiagPtr[lenA], 1*sizeof(unsigned int), hipMemcpyDeviceToHost);
		}
		unsigned int* tmp = d_prevprevDiagPtr;
		d_prevprevDiagPtr = d_prevDiagPtr;
		d_prevDiagPtr = d_currDiagPtr;
		d_currDiagPtr = tmp;
	}
//	cudaMemcpy((void*)currDiag, (void*)d_currDiagPtr, (lenA+1)*sizeof(unsigned int), cudaMemcpyDeviceToHost);

	t2 = chrono::high_resolution_clock::now();
	time_span = chrono::duration_cast<chrono::duration<double>>(t2 - t1);
	printf("\nelapsed time: %f\n", time_span.count());
	printf("edit distance: %d\n", ed);
	ed = 0;
	}
	return 0;
}
